#include "hip/hip_runtime.h"
/*********************************************************************[microsat.c]***

  The MIT License

  Copyright (c) 2014-2018 Marijn Heule

  Permission is hereby granted, free of charge, to any person obtaining a copy
  of this software and associated documentation files (the "Software"), to deal
  in the Software without restriction, including without limitation the rights
  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
  copies of the Software, and to permit persons to whom the Software is
  furnished to do so, subject to the following conditions:

  The above copyright notice and this permission notice shall be included in all
  copies or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
  SOFTWARE.

*************************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>

/**
 * Parameters
 */

// Specify how to compute the heuristic.
// #define NO_MRC
// #define MRC
// #define MRC_DYN
// #define MRC_GPU

// Specify the heuristic.
// #define JW_OS
// #define JW_TS
// #define BOHM
// #define POSIT
// #define DLIS
// #define DLCS
// #define RDLIS
// #define RDLCS

// Enable statistics.
// #define STATS

// #ifdef MRC_GPU
// #define NUM_THREADS_PER_BLOCK (512)
// #endif
#ifdef POSIT
#define POSIT_N (8)
#endif
#ifdef BOHM
#define BOHM_ALPHA (1)
#define BOHM_BETA (2)
#endif
#ifdef STATS
#define TIMEOUT (60)    // In s.
#endif

/**
 * End parameters
 */

#include "utils.cuh"
#include "sig_handling.h"
#ifdef MRC
#include "sat_miracle.cuh"
#endif
#ifdef MRC_DYN
#include "miracle_dynamic.cuh"
#endif
#ifdef MRC_GPU
#include "sat_miracle.cuh"
#include "launch_parameters_gpu.cuh"
#endif

#ifdef MRC_GPU
int num_threads_per_block;    // Number of threads per block.
static int *d_var_ass;        // d_mrc->var_ass on the host.
#endif

#ifdef POSIT
int POSIT_n;                  // Constant of the POSIT weight function.
#endif

#ifdef BOHM
int BOHM_alpha;               // Constant of the BOHM weight function.
int BOHM_beta;                // Constant of the BOHM weight function.
#endif

#if defined MRC || defined MRC_DYN || defined MRC_GPU
static Lit *lits;             // Array of assigned literals.
static int lits_len;          // Length of lits, which is the number of assigned literals.
#endif

#ifdef STATS
char* filename;               // Filename of the DIMACS CNF formula.
int mem_used;                 // The number of integers allocated in the DB
int nConflicts;               // Number of conflicts which is used to updates scores
int maxLemmas;                // Initial maximum number of learnt clauses
int nRestarts;                // The number of restarts performed

clock_t solve_tic;            // Solving time in clock.
clock_t solve_toc;            // Solving time out clock.
double solving_time;          // Solving time.

#if defined MRC || defined MRC_DYN || defined MRC_GPU
double miracle_time;          // MiraCle time.

clock_t inc_dec_lvl_tic;      // Decision level increase time in clock.
clock_t inc_dec_lvl_toc;      // Decision level increase time out clock.
double inc_dec_lvl_time;      // Decision level increase time.
double max_inc_dec_lvl_time;  // Maximum decision level increase time.
double min_inc_dec_lvl_time;  // Minimum decision level increase time.
double avg_inc_dec_lvl_time;  // Average decision level increase time.
double tot_inc_dec_lvl_time;  // Total decision level increase time.
int num_inc_dec_lvl;          // Number of increase decision level calls.
static int inc_dec_lvl_f;     // Flag for increase decision level.

clock_t assign_tic;           // Assignment time in clock.
clock_t assign_toc;           // Assignment time out clock.
double assign_time;           // Assignment time.
double max_assign_time;       // Maximum assignment time.
double min_assign_time;       // Minimum assignment time.
double avg_assign_time;       // Average assignment time.
double tot_assign_time;       // Total assignment time.
int num_assign;               // Number of assignment calls.
static int assign_f;          // Flag for assignment.

clock_t bj_tic;               // Backjumping time in clock.
clock_t bj_toc;               // Backjumping time out clock.
double bj_time;               // Backjumping time.
double max_bj_time;           // Maximum backjumping time.
double min_bj_time;           // Minimum backjumping time.
double avg_bj_time;           // Average backjumping time.
double tot_bj_time;           // Total backjumping time.
int num_bj;                   // Number of backjumping calls.
static int bj_f;              // Flag for backjumping.
#endif

clock_t heur_tic;             // Heuristic time in clock.
clock_t heur_toc;             // Heuristic time out clock.
double heur_time;             // Heuristic time.
double max_heur_time;         // Maximum heuristic time.
double min_heur_time;         // Minimum heuristic time.
double avg_heur_time;         // Average heuristic time.
double tot_heur_time;         // Total heuristic time.
int num_heur;                 // Number of heuristic calls.

int timeout_expired;          // Flag for timeout expiration.
int escape;                   // Flag for SIGINT.
int timeout;                  // In s.

void print_stats() {          // Print solving statistics
    printf("****************************************************************");
    printf("\n");
    printf("************************    STATS    ***************************");
    printf("\n");
    printf("****************************************************************");
    printf("\n\n");

    if (timeout_expired) {
        printf("Timeout expired: YES\n");
    } else {
        printf("Timeout expired: NO\n");
    }

    if (escape) {
        printf("SIGINT captured: YES\n");
    } else {
        printf("SIGINT captured: NO\n");
    }

    printf("Timeout: %d s\n", timeout);
#ifdef MRC_GPU
    printf("Number of threads per block: %d\n", gpu_num_threads_per_block());
#endif
#ifdef POSIT
    printf("POSIT n: %d\n", POSIT_n);
#endif
#ifdef BOHM
    printf("BOHM alpha: %d\n", BOHM_alpha);
    printf("BOHM beta: %d\n", BOHM_beta);
#endif
    printf("\n");

    printf("Solving time: %f ms\n", solving_time);
    printf("\n");

#if defined MRC || defined MRC_DYN || defined MRC_GPU
    printf("MiraCle time: %f ms\n", miracle_time);
    printf("%% of solving time used in MiraCle calls: %f %%\n",
           (miracle_time * 100) / solving_time);
    printf("\n");

    printf("Maximum decision level increase time: %f ms\n", max_inc_dec_lvl_time);
    printf("Minimum decision level increase time: %f ms\n", min_inc_dec_lvl_time);
    avg_inc_dec_lvl_time = tot_inc_dec_lvl_time / num_inc_dec_lvl;
    printf("Average decision level increase time: %f ms\n", avg_inc_dec_lvl_time);
    printf("Total decision level increase time: %f ms\n", tot_inc_dec_lvl_time);
    printf("%% of MiraCle time used in increase decision level calls: %f %%\n",
           (tot_inc_dec_lvl_time * 100) / miracle_time);
    printf("Number of increase decision level calls: %d\n", num_inc_dec_lvl);
    printf("\n");

    printf("Maximum assignment time: %f ms\n", max_assign_time);
    printf("Minimum assignment time: %f ms\n", min_assign_time);
    avg_assign_time = tot_assign_time / num_assign;
    printf("Average assignment time: %f ms\n", avg_assign_time);
    printf("Total assignment time: %f ms\n", tot_assign_time);
    printf("%% of MiraCle time used in assignment calls: %f %%\n",
           (tot_assign_time * 100) / miracle_time);
    printf("Number of assignment calls: %d\n", num_assign);
    printf("\n");

    printf("Maximum backjumping time: %f ms\n", max_bj_time);
    printf("Minimum backjumping time: %f ms\n", min_bj_time);
    avg_bj_time = tot_bj_time / num_bj;
    printf("Average backjumping time: %f ms\n", avg_bj_time);
    printf("Total backjumping time: %f ms\n", tot_bj_time);
    printf("%% of MiraCle time used in backjumping calls: %f %%\n",
           (tot_bj_time * 100) / miracle_time);
    printf("Number of backjumping calls: %d\n", num_bj);
    printf("\n");
#endif

    printf("Maximum heuristic time: %f ms\n", max_heur_time);
    printf("Minimum heuristic time: %f ms\n", min_heur_time);
    avg_heur_time = tot_heur_time / num_heur;
    printf("Average heuristic time: %f ms\n", avg_heur_time);
    printf("Total heuristic time: %f ms\n", tot_heur_time);
#if defined MRC || defined MRC_DYN || defined MRC_GPU
    printf("%% of MiraCle time used in heuristic calls: %f %%\n",
           (tot_heur_time * 100) / miracle_time);
#endif
#ifdef NO_MRC
    printf("%% of solving time used in heuristic calls: %f %%\n",
           (tot_heur_time * 100) / solving_time);
#endif
    printf("Number of heuristic calls: %d\n", num_heur);
    printf("\n");

    printf("****************************************************************");
    printf("\n");
    printf("**********************    END STATS    *************************");
    printf("\n");
    printf("****************************************************************");
    printf("\n\n");
}
#endif

enum { END = -9, UNSAT = 0, SAT = 1, MARK = 2, IMPLIED = 6, MEM_MAX = (1 << 30) };

struct solver { // The variables in the struct are described in the allocate procedure
  int  *DB, nVars, nClauses, mem_used, mem_fixed, maxLemmas, nLemmas, *buffer, nConflicts, *model,
       *reason, *falseStack, *falselit, *first, *forced, *processed, *assigned, *next, *prev, head, res, fast, slow, nRestarts; };

void unassign (struct solver* S, int lit) { S->falselit[lit] = 0; }   // Unassign the literal

void restart (struct solver* S) {                                     // Perform a restart (i.e., unassign all variables)
  while (S->assigned > S->forced) unassign (S, *(--S->assigned));     // Remove all unforced false lits from falseStack
  S->processed = S->forced; }                                         // Reset the processed pointer

void assign (struct solver* S, int* reason, int forced) {             // Make the first literal of the reason true
  int lit = reason[0];                                                // Let lit be the first literal in the reason
  S->falselit[-lit] = forced ? IMPLIED : 1;                           // Mark lit as true and IMPLIED if forced
  *(S->assigned++) = -lit;                                            // Push it on the assignment stack
  S->reason[abs (lit)] = 1 + (int) ((reason)-S->DB);                  // Set the reason clause of lit
  S->model [abs (lit)] = (lit > 0); }                                 // Mark the literal as true in the model

void addWatch (struct solver* S, int lit, int mem) {                  // Add a watch pointer to a clause containing lit
  S->DB[mem] = S->first[lit]; S->first[lit] = mem; }                  // By updating the database and the pointers

int* getMemory (struct solver* S, int mem_size) {                     // Allocate memory of size mem_size
  if (S->mem_used > MEM_MAX - mem_size) {                             // In case the code is used within a code base
    printf ("c out of memory\n"); exit (1); }
  int *store = (S->DB + S->mem_used);                                 // Compute a pointer to the new memory location
  S->mem_used += mem_size;                                            // Update the size of the used memory
#ifdef STATS
  mem_used = S->mem_used;
#endif
  return store; }                                                     // Return the pointer

int* addClause (struct solver* S, int* in, int size, int irr) {       // Adds a clause stored in *in of size size
  int i, used = S->mem_used;                                          // Store a pointer to the beginning of the clause
  int* clause = getMemory (S, size + 3) + 2;                          // Allocate memory for the clause in the database
  if (size >  1) { addWatch (S, in[0], used  );                       // If the clause is not unit, then add
                   addWatch (S, in[1], used+1); }                     // Two watch pointers to the datastructure
  for (i = 0; i < size; i++) clause[i] = in[i]; clause[i] = 0;        // Copy the clause from the buffer to the database
  if (irr) S->mem_fixed = S->mem_used; else S->nLemmas++;             // Update the statistics
  return clause; }                                                    // Return the pointer to the clause in the database

void reduceDB (struct solver* S, int k) {                     // Removes "less useful" lemmas from DB
  while (S->nLemmas > S->maxLemmas) S->maxLemmas += 300;      // Allow more lemmas in the future
#ifdef STATS
  maxLemmas = S->maxLemmas;
#endif
  S->nLemmas = 0;                                             // Reset the number of lemmas

  int i; for (i = -S->nVars; i <= S->nVars; i++) {            // Loop over the variables
    if (i == 0) continue; int* watch = &S->first[i];          // Get the pointer to the first watched clause
    while (*watch != END)                                     // As long as there are watched clauses
      if (*watch < S->mem_fixed) watch = (S->DB + *watch);    // Remove the watch if it points to a lemma
      else                      *watch =  S->DB[  *watch]; }  // Otherwise (meaning an input clause) go to next watch

  int old_used = S->mem_used; S->mem_used = S->mem_fixed;     // Virtually remove all lemmas
#ifdef STATS
  mem_used = S->mem_used;
#endif
  for (i = S->mem_fixed + 2; i < old_used; i += 3) {          // While the old memory contains lemmas
    int count = 0, head = i;                                  // Get the lemma to which the head is pointing
    while (S->DB[i]) { int lit = S->DB[i++];                  // Count the number of literals
      if ((lit > 0) == S->model[abs (lit)]) count++; }        // That are satisfied by the current model
    if (count < k) addClause (S, S->DB+head, i-head, 0); } }  // If the latter is smaller than k, add it back

void bump (struct solver* S, int lit) {                       // Move the variable to the front of the decision list
  if (S->falselit[lit] != IMPLIED) { S->falselit[lit] = MARK; // MARK the literal as involved if not a top-level unit
#ifdef NO_MRC
    int var = abs (lit); if (var != S->head) {                // In case var is not already the head of the list
      S->prev[S->next[var]] = S->prev[var];                   // Update the prev link, and
      S->next[S->prev[var]] = S->next[var];                   // Update the next link, and
      S->next[S->head] = var;                                 // Add a next link to the head, and
      S->prev[var] = S->head; S->head = var; }                // Make var the new head
#endif
  }
}

int implied (struct solver* S, int lit) {                           // Check if lit(eral) is implied by MARK literals
  if (S->falselit[lit] > MARK) return (S->falselit[lit] & MARK);    // If checked before return old result
  if (!S->reason[abs (lit)]) return 0;                              // In case lit is a decision, it is not implied
  int* p = (S->DB + S->reason[abs (lit)] - 1);                      // Get the reason of lit(eral)
  while (*(++p))                                                    // While there are literals in the reason
    if ((S->falselit[*p] ^ MARK) && !implied (S, *p)) {             // Recursively check if non-MARK literals are implied
      S->falselit[lit] = IMPLIED - 1; return 0; }                   // Mark and return not implied (denoted by IMPLIED - 1)
  S->falselit[lit] = IMPLIED; return 1; }                           // Mark and return that the literal is implied

int* analyze (struct solver* S, int* clause) {            // Compute a resolvent from falsified clause
  S->res++; S->nConflicts++;                              // Bump restarts and update the statistic
#ifdef STATS
  nConflicts = S->nConflicts;
#endif
  while (*clause) bump (S, *(clause++));                  // MARK all literals in the falsified clause
  while (S->reason[abs (*(--S->assigned))]) {             // Loop on variables on falseStack until the last decision
    if (S->falselit[*S->assigned] == MARK) {              // If the tail of the stack is MARK
      int *check = S->assigned;                           // Pointer to check if first-UIP is reached
      while (S->falselit[*(--check)] != MARK)             // Check for a MARK literal before decision
        if (!S->reason[abs(*check)]) goto build;          // Otherwise it is the first-UIP so break
      clause = S->DB + S->reason[abs (*S->assigned)];     // Get the reason and ignore first literal
      while (*clause) bump (S, *(clause++)); }            // MARK all literals in reason
    unassign (S, *S->assigned); }                         // Unassign the tail of the stack

  build:; int size = 0, lbd = 0, flag = 0;                // Build conflict clause; Empty the clause buffer
  int* p = S->processed = S->assigned;                    // Loop from tail to front
  while (p >= S->forced) {                                // Only literals on the stack can be MARKed
    if ((S->falselit[*p] == MARK) && !implied (S, *p)) {  // If MARKed and not implied
      S->buffer[size++] = *p; flag = 1; }                 // Add literal to conflict clause buffer
    if (!S->reason[abs (*p)]) { lbd += flag; flag = 0;    // Increase LBD for a decision with a true flag
      if (size == 1) S->processed = p; }                  // And update the processed pointer
    S->falselit[*(p--)] = 1; }                            // Reset the MARK flag for all variables on the stack

  S->fast -= S->fast >>  5; S->fast += lbd << 15;      // Update the fast moving average
  S->slow -= S->slow >> 15; S->slow += lbd <<  5;      // Update the slow moving average

  while (S->assigned > S->processed)                   // Loop over all unprocessed literals
    unassign (S, *(S->assigned--));                    // Unassign all lits between tail & head
  unassign (S, *S->assigned);                          // Assigned now equal to processed
  S->buffer[size] = 0;                                 // Terminate the buffer (and potentially print clause)
  return addClause (S, S->buffer, size, 0); }          // Add new conflict clause to redundant DB

int propagate (struct solver* S) {                  // Performs unit propagation
  int forced = S->reason[abs (*S->processed)];      // Initialize forced flag
  while (S->processed < S->assigned) {              // While unprocessed false literals
    int lit = *(S->processed++);                    // Get first unprocessed literal
    int* watch = &S->first[lit];                    // Obtain the first watch pointer
    while (*watch != END) {                         // While there are watched clauses (watched by lit)
      int i, unit = 1;                              // Let's assume that the clause is unit
      int* clause = (S->DB + *watch + 1);	          // Get the clause from DB
      if (clause[-2] ==   0) clause++;              // Set the pointer to the first literal in the clause
      if (clause[ 0] == lit) clause[0] = clause[1]; // Ensure that the other watched literal is in front
      for (i = 2; unit && clause[i]; i++)           // Scan the non-watched literals
        if (!S->falselit[clause[i]]) {              // When clause[i] is not false, it is either true or unset
          clause[1] = clause[i]; clause[i] = lit;   // Swap literals
          int store = *watch; unit = 0;             // Store the old watch
          *watch = S->DB[*watch];                   // Remove the watch from the list of lit
          addWatch (S, clause[1], store); }         // Add the watch to the list of clause[1]
      if (unit) {                                   // If the clause is indeed unit
        clause[1] = lit; watch = (S->DB + *watch);  // Place lit at clause[1] and update next watch
        if ( S->falselit[-clause[0]]) continue;     // If the other watched literal is satisfied continue
        if (!S->falselit[ clause[0]]) {             // If the other watched literal is falsified,
          assign (S, clause, forced); }             // A unit clause is found, and the reason is set
        else { if (forced) return UNSAT;            // Found a root level conflict -> UNSAT
          int* lemma = analyze (S, clause);	        // Analyze the conflict return a conflict clause
          if (!lemma[1]) forced = 1;                // In case a unit clause is found, set forced flag
          assign (S, lemma, forced); break; } } } } // Assign the conflict clause as a unit
  if (forced) S->forced = S->processed;	            // Set S->forced if applicable
  return SAT; }	                                    // Finally, no conflict was found

#ifdef NO_MRC
int solve (struct solver* S) {                                      // Determine satisfiability
#endif
#ifdef MRC
int solve (struct solver* S, SAT_Miracle *sat_mrc) {
#endif
#ifdef MRC_DYN
int solve (struct solver* S, Miracle_Dyn *mrc_dyn) {
#endif
#ifdef MRC_GPU
int solve (struct solver* S, SAT_Miracle *sat_mrc) {
#endif

#if defined MRC || defined MRC_DYN || defined MRC_GPU
  int *last_dec;
  Lit lit;
  Var last_bvar;
  int last_bvar_ass;
  int decision;
#endif

#ifdef NO_MRC
  int decision = S->head;                                           // Initialize the solver
#endif
  S->res = 0;
  for (;;) {                                                        // Main solve loop

#ifdef STATS
    if (timeout_expired || escape) {
      exit(EXIT_SUCCESS);
    }
#endif

    int old_nLemmas = S->nLemmas;                                   // Store nLemmas to see whether propagate adds lemmas
    if (propagate (S) == UNSAT) return UNSAT;                       // Propagation returns UNSAT for a root level conflict

    if (S->nLemmas > old_nLemmas) {                                 // If the last decision caused a conflict
#ifdef NO_MRC
      decision = S->head;                                           // Reset the decision heuristic to head
#endif
      if (S->fast > (S->slow / 100) * 125) {                        // If fast average is substantially larger than slow average
//        printf("c restarting after %i conflicts (%i %i) %i\n", S->res, S->fast, S->slow, S->nLemmas > S->maxLemmas);
        S->res = 0; S->fast = (S->slow / 100) * 125;                // Restart and update the averages
        restart (S); S->nRestarts++;
#ifdef STATS
        nRestarts = S->nRestarts;
#endif

#if defined MRC || defined MRC_DYN || defined MRC_GPU
        last_dec = S->forced - 1;
        lits_len = 0;

        while (last_dec >= S->falseStack) {
          lit = neg_lit(*last_dec);
          last_dec--;
          lits[lits_len] = lit;
          lits_len++;
        }
#endif
#ifdef MRC
#ifdef STATS
        bj_tic = clock();
#endif
        mrc_backjump(0, sat_mrc);
#ifdef STATS
        bj_toc = clock();
        bj_f = 1;
#endif

        if (lits_len > 0) {
#ifdef STATS
          assign_tic = clock();
#endif
          mrc_assign_lits(lits, lits_len, sat_mrc);
#ifdef STATS
          assign_toc = clock();
          assign_f = 1;
#endif
        }
#endif
#ifdef MRC_DYN
#ifdef STATS
        bj_tic = clock();
#endif
        mrc_dyn_backjump(0, mrc_dyn);
#ifdef STATS
        bj_toc = clock();
        bj_f = 1;
#endif

        if (lits_len > 0) {
#ifdef STATS
          assign_tic = clock();
#endif
          mrc_dyn_assign_lits(lits, lits_len, mrc_dyn);
#ifdef STATS
          assign_toc = clock();
          assign_f = 1;
#endif
        }
#endif
#ifdef MRC_GPU
#ifdef STATS
        bj_tic = clock();
#endif
        mrc_gpu_backjump(0, sat_mrc);
#ifdef STATS
        bj_toc = clock();
        bj_f = 1;
#endif

        if (lits_len > 0) {
#ifdef STATS
          assign_tic = clock();
#endif
          mrc_gpu_assign_lits(lits, lits_len, sat_mrc);
#ifdef STATS
          assign_toc = clock();
          assign_f = 1;
#endif
        }
#endif
#if defined STATS && (defined MRC || defined MRC_DYN || defined MRC_GPU)
        if (bj_f) {
          num_bj++;
          bj_time = ((double)(bj_toc - bj_tic)) / CLOCKS_PER_SEC;   // In s.
          bj_time *= 1000;    // In ms.

          tot_bj_time += bj_time;
          miracle_time += bj_time;

          if (bj_time > max_bj_time) {
            max_bj_time = bj_time;
          }

          if (bj_time < min_bj_time) {
            min_bj_time = bj_time;
          }

          bj_f = 0;
        }

        if (assign_f) {
          num_assign++;
          assign_time = ((double)(assign_toc - assign_tic)) / CLOCKS_PER_SEC;   // In s.
          assign_time *= 1000;    // In ms.

          tot_assign_time += assign_time;
          miracle_time += assign_time;

          if (assign_time > max_assign_time) {
            max_assign_time = assign_time;
          }

          if (assign_time < min_assign_time) {
            min_assign_time = assign_time;
          }

          assign_f = 0;
        }
#endif

        if (S->nLemmas > S->maxLemmas) reduceDB (S, 6); } }         // Reduce the DB when it contains too many lemmas

#if defined MRC || defined MRC_DYN || defined MRC_GPU
    last_dec = S->assigned - 1;
    lits_len = 0;
    last_bvar = UNDEF_VAR;

    while (last_dec >= S->falseStack) {
      lit = neg_lit(*last_dec);
      last_dec--;
      lits[lits_len] = lit;
      lits_len++;

      if (S->reason[abs(lit)] == 0) {
        last_bvar = lit_to_var(lit);
        break;
      }
    }
#endif
#ifdef MRC
    if (lits_len > 0) {
      if (last_bvar == UNDEF_VAR) {
#ifdef STATS
        bj_tic = clock();
#endif
        mrc_backjump(0, sat_mrc);
#ifdef STATS
        bj_toc = clock();
        bj_f = 1;
#endif
#ifdef STATS
        assign_tic = clock();
#endif
        mrc_assign_lits(lits, lits_len, sat_mrc);
#ifdef STATS
        assign_toc = clock();
        assign_f = 1;
#endif
      } else {
        last_bvar_ass = abs(sat_mrc->mrc->var_ass[last_bvar]);

        if (last_bvar_ass) {
#ifdef STATS
          bj_tic = clock();
#endif
          mrc_backjump(last_bvar_ass - 1, sat_mrc);
#ifdef STATS
          bj_toc = clock();
          bj_f = 1;
#endif
#ifdef STATS
          inc_dec_lvl_tic = clock();
#endif
          mrc_increase_decision_level(sat_mrc);
#ifdef STATS
          inc_dec_lvl_toc = clock();
          inc_dec_lvl_f = 1;
#endif
#ifdef STATS
          assign_tic = clock();
#endif
          mrc_assign_lits(lits, lits_len, sat_mrc);
#ifdef STATS
          assign_toc = clock();
          assign_f = 1;
#endif
        } else {
#ifdef STATS
          inc_dec_lvl_tic = clock();
#endif
          mrc_increase_decision_level(sat_mrc);
#ifdef STATS
          inc_dec_lvl_toc = clock();
          inc_dec_lvl_f = 1;
#endif
#ifdef STATS
          assign_tic = clock();
#endif
          mrc_assign_lits(lits, lits_len, sat_mrc);
#ifdef STATS
          assign_toc = clock();
          assign_f = 1;
#endif
        }
      }
    }
#endif
#ifdef MRC_DYN
    if (lits_len > 0) {
      if (last_bvar == UNDEF_VAR) {
#ifdef STATS
        bj_tic = clock();
#endif
        mrc_dyn_backjump(0, mrc_dyn);
#ifdef STATS
        bj_toc = clock();
        bj_f = 1;
#endif
#ifdef STATS
        assign_tic = clock();
#endif
        mrc_dyn_assign_lits(lits, lits_len, mrc_dyn);
#ifdef STATS
        assign_toc = clock();
        assign_f = 1;
#endif
      } else {
        last_bvar_ass = abs(mrc_dyn->var_ass[last_bvar]);

        if (last_bvar_ass) {
#ifdef STATS
          bj_tic = clock();
#endif
          mrc_dyn_backjump(last_bvar_ass - 1, mrc_dyn);
#ifdef STATS
          bj_toc = clock();
          bj_f = 1;
#endif
#ifdef STATS
          inc_dec_lvl_tic = clock();
#endif
          mrc_dyn_increase_decision_level(mrc_dyn);
#ifdef STATS
          inc_dec_lvl_toc = clock();
          inc_dec_lvl_f = 1;
#endif
#ifdef STATS
          assign_tic = clock();
#endif
          mrc_dyn_assign_lits(lits, lits_len, mrc_dyn);
#ifdef STATS
          assign_toc = clock();
          assign_f = 1;
#endif
        } else {
#ifdef STATS
          inc_dec_lvl_tic = clock();
#endif
          mrc_dyn_increase_decision_level(mrc_dyn);
#ifdef STATS
          inc_dec_lvl_toc = clock();
          inc_dec_lvl_f = 1;
#endif
#ifdef STATS
          assign_tic = clock();
#endif
          mrc_dyn_assign_lits(lits, lits_len, mrc_dyn);
#ifdef STATS
          assign_toc = clock();
          assign_f = 1;
#endif
        }
      }
    }
#endif
#ifdef MRC_GPU
    if (lits_len > 0) {
      if (last_bvar == UNDEF_VAR) {
#ifdef STATS
        bj_tic = clock();
#endif
        mrc_gpu_backjump(0, sat_mrc);
#ifdef STATS
        bj_toc = clock();
        bj_f = 1;
#endif
#ifdef STATS
        assign_tic = clock();
#endif
        mrc_gpu_assign_lits(lits, lits_len, sat_mrc);
#ifdef STATS
        assign_toc = clock();
        assign_f = 1;
#endif
      } else {
        gpuErrchk( hipMemcpy(&last_bvar_ass, &(d_var_ass[last_bvar]),
                              sizeof last_bvar_ass,
                              hipMemcpyDeviceToHost) );

        last_bvar_ass = abs(last_bvar_ass);

        if (last_bvar_ass) {
#ifdef STATS
          bj_tic = clock();
#endif
          mrc_gpu_backjump(last_bvar_ass - 1, sat_mrc);
#ifdef STATS
          bj_toc = clock();
          bj_f = 1;
#endif
#ifdef STATS
          inc_dec_lvl_tic = clock();
#endif
          mrc_gpu_increase_decision_level(sat_mrc);
#ifdef STATS
          inc_dec_lvl_toc = clock();
          inc_dec_lvl_f = 1;
#endif
#ifdef STATS
          assign_tic = clock();
#endif
          mrc_gpu_assign_lits(lits, lits_len, sat_mrc);
#ifdef STATS
          assign_toc = clock();
          assign_f = 1;
#endif
        } else {
#ifdef STATS
          inc_dec_lvl_tic = clock();
#endif
          mrc_gpu_increase_decision_level(sat_mrc);
#ifdef STATS
          inc_dec_lvl_toc = clock();
          inc_dec_lvl_f = 1;
#endif
#ifdef STATS
          assign_tic = clock();
#endif
          mrc_gpu_assign_lits(lits, lits_len, sat_mrc);
#ifdef STATS
          assign_toc = clock();
          assign_f = 1;
#endif
        }
      }
    }
#endif
#if defined STATS && (defined MRC || defined MRC_DYN || defined MRC_GPU)
  if (bj_f) {
    num_bj++;
    bj_time = ((double)(bj_toc - bj_tic)) / CLOCKS_PER_SEC;   // In s.
    bj_time *= 1000;    // In ms.

    tot_bj_time += bj_time;
    miracle_time += bj_time;

    if (bj_time > max_bj_time) {
      max_bj_time = bj_time;
    }

    if (bj_time < min_bj_time) {
      min_bj_time = bj_time;
    }

    bj_f = 0;
  }

  if (inc_dec_lvl_f) {
    num_inc_dec_lvl++;
    inc_dec_lvl_time = ((double)(inc_dec_lvl_toc - inc_dec_lvl_tic)) / CLOCKS_PER_SEC;  // In s.
    inc_dec_lvl_time *= 1000;   // In ms.

    tot_inc_dec_lvl_time += inc_dec_lvl_time;
    miracle_time += inc_dec_lvl_time;
    
    if (inc_dec_lvl_time > max_inc_dec_lvl_time) {
      max_inc_dec_lvl_time = inc_dec_lvl_time;
    }

    if (inc_dec_lvl_time < min_inc_dec_lvl_time) {
      min_inc_dec_lvl_time = inc_dec_lvl_time;
    }

    inc_dec_lvl_f = 0;
  }

  if (assign_f) {
    num_assign++;
    assign_time = ((double)(assign_toc - assign_tic)) / CLOCKS_PER_SEC;   // In s.
    assign_time *= 1000;    // In ms.

    tot_assign_time += assign_time;
    miracle_time += assign_time;

    if (assign_time > max_assign_time) {
      max_assign_time = assign_time;
    }

    if (assign_time < min_assign_time) {
      min_assign_time = assign_time;
    }

    assign_f = 0;
  }
#endif

#ifdef MRC
#ifdef STATS
    heur_tic = clock();
#endif
    #ifdef JW_OS
    decision = mrc_JW_OS_heuristic(sat_mrc);
    #endif
    #ifdef JW_TS
    decision = mrc_JW_TS_heuristic(sat_mrc);
    #endif
    #ifdef BOHM
    decision = mrc_BOHM_heuristic(sat_mrc, BOHM_alpha, BOHM_beta);
    #endif
    #ifdef POSIT
    decision = mrc_POSIT_heuristic(sat_mrc, POSIT_n);
    #endif
    #ifdef DLIS
    decision = mrc_DLIS_heuristic(sat_mrc);
    #endif
    #ifdef DLCS
    decision = mrc_DLCS_heuristic(sat_mrc);
    #endif
    #ifdef RDLIS
    decision = mrc_RDLIS_heuristic(sat_mrc);
    #endif
    #ifdef RDLCS
    decision = mrc_RDLCS_heuristic(sat_mrc);
    #endif
#ifdef STATS
    heur_toc = clock();
#endif
#endif
#ifdef MRC_DYN
#ifdef STATS
    heur_tic = clock();
#endif
    #ifdef JW_OS
    decision = mrc_dyn_JW_OS_heuristic(mrc_dyn);
    #endif
    #ifdef JW_TS
    decision = mrc_dyn_JW_TS_heuristic(mrc_dyn);
    #endif
    #ifdef BOHM
    decision = mrc_dyn_BOHM_heuristic(mrc_dyn, BOHM_alpha, BOHM_beta);
    #endif
    #ifdef POSIT
    decision = mrc_dyn_POSIT_heuristic(mrc_dyn, POSIT_n);
    #endif
    #ifdef DLIS
    decision = mrc_dyn_DLIS_heuristic(mrc_dyn);
    #endif
    #ifdef DLCS
    decision = mrc_dyn_DLCS_heuristic(mrc_dyn);
    #endif
    #ifdef RDLIS
    decision = mrc_dyn_RDLIS_heuristic(mrc_dyn);
    #endif
    #ifdef RDLCS
    decision = mrc_dyn_RDLCS_heuristic(mrc_dyn);
    #endif
#ifdef STATS
    heur_toc = clock();
#endif
#endif
#ifdef MRC_GPU
#ifdef STATS
    heur_tic = clock();
#endif
    #ifdef JW_OS
    decision = mrc_gpu_JW_OS_heuristic(sat_mrc);
    #endif
    #ifdef JW_TS
    decision = mrc_gpu_JW_TS_heuristic(sat_mrc);
    #endif
    #ifdef BOHM
    decision = mrc_gpu_BOHM_heuristic(sat_mrc, BOHM_alpha, BOHM_beta);
    #endif
    #ifdef POSIT
    decision = mrc_gpu_POSIT_heuristic(sat_mrc, POSIT_n);
    #endif
    #ifdef DLIS
    decision = mrc_gpu_DLIS_heuristic(sat_mrc);
    #endif
    #ifdef DLCS
    decision = mrc_gpu_DLCS_heuristic(sat_mrc);
    #endif
    #ifdef RDLIS
    decision = mrc_gpu_RDLIS_heuristic(sat_mrc);
    #endif
    #ifdef RDLCS
    decision = mrc_gpu_RDLCS_heuristic(sat_mrc);
    #endif
#ifdef STATS
    heur_toc = clock();
#endif
#endif

#ifdef NO_MRC
#ifdef STATS
    heur_tic = clock();
#endif
    while (S->falselit[decision] || S->falselit[-decision]) {       // As long as the temporary decision is assigned
      decision = S->prev[decision]; }                               // Replace it with the next variable in the decision list
    decision = S->model[decision] ? decision : -decision;           // Assign the decision variable based on the model
#ifdef STATS
    heur_toc = clock();
#endif
    if (decision == 0) return SAT;                                  // If the end of the list is reached, then a solution is found
    S->falselit[-decision] = 1;                                     // Assign the decision literal to true (change to IMPLIED-1?)
    *(S->assigned++) = -decision;                                   // And push it on the assigned stack
    decision = abs(decision); S->reason[decision] = 0;              // Decisions have no reason clauses
#endif
#if defined MRC || defined MRC_DYN || defined MRC_GPU
    if (decision == UNDEF_LIT) {
      return SAT;
    }

    S->falselit[-decision] = 1;
    *(S->assigned++) = -decision;
    S->model[abs(decision)] = decision > 0 ? 1 : 0;
    decision = abs(decision); S->reason[decision] = 0;
#endif

#ifdef STATS
    num_heur++;
    heur_time = ((double)(heur_toc - heur_tic)) / CLOCKS_PER_SEC;    // In s.
    heur_time *= 1000;   // In ms.

    tot_heur_time += heur_time;
#if defined MRC || defined MRC_DYN || defined MRC_GPU
    miracle_time += heur_time;
#endif

    if (heur_time > max_heur_time) {
      max_heur_time = heur_time;
    }

    if (heur_time < min_heur_time) {
      min_heur_time = heur_time;
    }
#endif
  }
}

void initCDCL (struct solver* S, int n, int m) {
  if (n < 1)      n = 1;                  // The code assumes that there is at least one variable
  S->nVars          = n;                  // Set the number of variables
  S->nClauses       = m;                  // Set the number of clauases
  S->mem_used       = 0;                  // The number of integers allocated in the DB
#ifdef STATS
  mem_used = S->mem_used;
#endif
  S->nLemmas        = 0;                  // The number of learned clauses -- redundant means learned
  S->nConflicts     = 0;                  // Number of conflicts which is used to updates scores
#ifdef STATS
  nConflicts = S->nConflicts;
#endif
  S->maxLemmas      = 2000;               // Initial maximum number of learnt clauses
#ifdef STATS
  maxLemmas = S->maxLemmas;
#endif
  S->fast = S->slow = 1 << 24;            // Initialize the fast and slow moving averages
  S->nRestarts      = 0;                  // The number of restarts performed
#ifdef STATS
  nRestarts = S->nRestarts;
#endif

  S->DB = (int *) malloc (sizeof (int) * MEM_MAX); // Allocate the initial database
  S->model       = getMemory (S, n+1); // Full assignment of the (Boolean) variables (initially set to false)
#ifdef NO_MRC
  S->next        = getMemory (S, n+1); // Next variable in the heuristic order
  S->prev        = getMemory (S, n+1); // Previous variable in the heuristic order
#endif
  S->buffer      = getMemory (S, n  ); // A buffer to store a temporary clause
  S->reason      = getMemory (S, n+1); // Array of clauses
  S->falseStack  = getMemory (S, n+1); // Stack of falsified literals -- this pointer is never changed
  S->forced      = S->falseStack;      // Points inside *falseStack at first decision (unforced literal)
  S->processed   = S->falseStack;      // Points inside *falseStack at first unprocessed literal
  S->assigned    = S->falseStack;      // Points inside *falseStack at last unprocessed literal
  S->falselit    = getMemory (S, 2*n+1); S->falselit += n; // Labels for variables, non-zero means false
  S->first       = getMemory (S, 2*n+1); S->first += n; // Offset of the first watched clause
  S->DB[S->mem_used++] = 0;            // Make sure there is a 0 before the clauses are loaded.
#ifdef STATS
  mem_used = S->mem_used;
#endif

  int i; for (i = 1; i <= n; i++) {                        // Initialize the main datastructures:
#ifdef NO_MRC
    S->prev [i] = i - 1; S->next[i-1] = i;                 // the double-linked list for variable-move-to-front,
#endif
    S->model[i] = S->falselit[-i] = S->falselit[i] = 0;    // the model (phase-saving), the falselit array,
    S->first[i] = S->first[-i] = END; }                    // and first (watch pointers).
#ifdef NO_MRC
  S->head = n;                                             // Initialize the head of the double-linked list
#endif
}

static void read_until_new_line (FILE * input) {
  int ch;
  while ((ch = getc (input)) != '\n')
    if (ch == EOF) { printf ("parse error: unexpected EOF"); exit (1); }
}

int parse (struct solver* S, char* filename) {                            // Parse the formula and initialize
  int tmp; FILE* input; int close = 1;
  if (strcmp (filename + strlen (filename) - 3, ".xz"))
    input = fopen (filename, "r");					  // Open file
  else { char * cmd = (char *)malloc (strlen (filename) + 20);
    sprintf (cmd, "xz -c -d %s", filename);
    input = popen (cmd, "r"); close = 2; free (cmd); }		          // Open pipe
  while ((tmp = getc (input)) == 'c')
    read_until_new_line (input);
  ungetc (tmp, input);
  do { tmp = fscanf (input, "p cnf %d %d", &S->nVars, &S->nClauses);  // Find the first non-comment line
    if (tmp > 0 && tmp != EOF) break; tmp = fscanf (input, "%*s\n"); }    // In case a commment line was found
  while (tmp != 2 && tmp != EOF);                                         // Skip it and read next line

  initCDCL (S, S->nVars, S->nClauses);                     // Allocate the main datastructures
  int nZeros = S->nClauses, size = 0;                      // Initialize the number of clauses to read
  while (nZeros > 0) {                                     // While there are clauses in the file
    int ch = getc (input);
    if (ch == ' ' || ch == '\n') continue;
    if (ch == 'c') { read_until_new_line (input); continue; }
    ungetc (ch, input);
    int lit = 0; tmp = fscanf (input, " %i ", &lit);          // Read a literal.
    if (!lit) {                                               // If reaching the end of the clause
      int* clause = addClause (S, S->buffer, size, 1);        // Then add the clause to data_base
      if (!size || ((size == 1) && S->falselit[clause[0]]))   // Check for empty clause or conflicting unit
        return UNSAT;                                         // If either is found return UNSAT
      if ((size == 1) && !S->falselit[-clause[0]]) {          // Check for a new unit
        assign (S, clause, 1); }                              // Directly assign new units (forced = 1)
      size = 0; --nZeros; }                                   // Reset buffer
    else S->buffer[size++] = lit; }                           // Add literal to buffer
  if (close == 1) fclose (input);                             // Close the formula file
  if (close == 2) pclose (input);                             // Close the formula pipe
  return SAT; }                                               // Return that no conflict was observed

/**
 * Display the result of the solver
 */
void show_result(struct solver* S, int result) {
  if (result == SAT) {    // if the formula is satisfiable
    printf("SAT\n");

    for (int v = 1; v <= S->nVars; v++) {
      if (S->model[v] == 0) {
        printf("%d ", -v);
      } else {
        printf("%d ", v);
      }
    }

    printf("0\n");
  } else {                // if the formula is unsatisfiable
    printf("UNSAT\n");
  }
}

int main (int argc, char** argv) {			                      // The main procedure for a STANDALONE solver
  if (argc < 2) abort ();

#ifdef STATS
  filename = argv[1];
#endif
  
  struct solver S;	                                          // Create the solver datastructure

  if (parse (&S, argv[1]) == UNSAT) {                         // Parse the DIMACS file in argv[1]
    show_result(&S, UNSAT);
  } else {
#ifdef POSIT
    POSIT_n = POSIT_N;
#endif
#ifdef BOHM
    BOHM_alpha = BOHM_ALPHA;
    BOHM_beta = BOHM_BETA;
#endif
#if defined MRC || defined MRC_DYN || defined MRC_GPU
    lits = (Lit *)malloc(sizeof *lits * S.nVars);
    lits_len = 0;
#endif
#ifdef STATS
#if defined MRC || defined MRC_DYN || defined MRC_GPU
    miracle_time = 0;

    max_inc_dec_lvl_time = -DBL_MAX;
    min_inc_dec_lvl_time = DBL_MAX;
    tot_inc_dec_lvl_time = 0;
    num_inc_dec_lvl = 0;
    inc_dec_lvl_f = 0;

    max_assign_time = -DBL_MAX;
    min_assign_time = DBL_MAX;
    tot_assign_time = 0;
    num_assign = 0;
    assign_f = 0;

    max_bj_time = -DBL_MAX;
    min_bj_time = DBL_MAX;
    tot_bj_time = 0;
    num_bj = 0;
    bj_f = 0;
#endif

    max_heur_time = -DBL_MAX;
    min_heur_time = DBL_MAX;
    tot_heur_time = 0;
    num_heur = 0;

    timeout_expired = 0;
    escape = 0;
    timeout = TIMEOUT;   // In s.

    // Set SIGINT handler.
    install_handler();

    // Set SIGALRM handler.
    install_alarmhandler();
#endif

#ifdef NO_MRC
#ifdef STATS
    alarm(timeout);
    solve_tic = clock();
#endif
    int slv = solve (&S);                                       // Solve without limit (number of conflicts)
#ifdef STATS
    solve_toc = clock();
#endif
#endif
#ifdef MRC
    SAT_Miracle *sat_mrc = mrc_create_sat_miracle(argv[1], false);

#ifdef STATS
    alarm(timeout);
    solve_tic = clock();
#endif
    int slv = solve (&S, sat_mrc);
#ifdef STATS
    solve_toc = clock();
#endif

    mrc_destroy_sat_miracle(sat_mrc);
#endif
#ifdef MRC_DYN
    Miracle_Dyn *mrc_dyn = mrc_dyn_create_miracle(argv[1]);

#ifdef STATS
    alarm(timeout);
    solve_tic = clock();
#endif
    int slv = solve (&S, mrc_dyn);
#ifdef STATS
    solve_toc = clock();
#endif

    mrc_dyn_destroy_miracle(mrc_dyn);
#endif
#ifdef MRC_GPU
    num_threads_per_block = NUM_THREADS_PER_BLOCK;
    gpu_set_device(0);
    gpu_set_num_threads_per_block(num_threads_per_block);

    SAT_Miracle *sat_mrc = mrc_create_sat_miracle(argv[1], true);

    gpuErrchk( hipMemcpy(&d_var_ass, &(sat_mrc->d_mrc->var_ass),
                          sizeof d_var_ass,
                          hipMemcpyDeviceToHost) );

#ifdef STATS
    alarm(timeout);
    solve_tic = clock();
#endif
    int slv = solve (&S, sat_mrc);
#ifdef STATS
    solve_toc = clock();
#endif

    mrc_destroy_sat_miracle(sat_mrc);
#endif

    if (slv == UNSAT) {                                         // Print whether the formula has a solution
      show_result(&S, UNSAT);
    } else {
      show_result(&S, SAT);
    }
  }

  printf ("\n");
  printf ("c statistics of %s: mem: %i conflicts: %i max_lemmas: %i restarts: %i\n", argv[1], S.mem_used, S.nConflicts, S.maxLemmas, S.nRestarts);

#ifdef STATS
  solving_time = ((double)(solve_toc - solve_tic)) / CLOCKS_PER_SEC;    // In s.
  solving_time *= 1000;   // In ms.

  printf("\n");
  print_stats();
#endif

  return 0;
}
